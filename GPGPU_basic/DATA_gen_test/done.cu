#include "hip/hip_runtime.h"

/**
* Toy example porting a simd algorithm to CUDA for an NVIDIA GPU
*
* Calculates the average size of set of 3d vectors.
* Example input: {{1,1,1},{2,2,2}}
* I.e., 1 vector of size (3*1^2)^-0.5 and 1 vector of size (3*2^2)^0.5
* Example output: ( sqrt(3) + sqrt(12) ) / 2 = 2.59808
*/
#include <iostream>
#include <vector>
#include <chrono>     // timing library
#include <numeric>    // std::accumulate()

#include "test-data.hpp"
// #include "data-sanity-check.hpp"




__host__ __device__
bool dom(XY a, XY b){
   if( ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y))){
      return true; 
   }
   return false;
}


__global__ 
void solv(int n, XY *input, int *output)
{
// Printing input
   // int index = threadIdx.x + blockIdx.x*(1024) ;
   int index = threadIdx.x + blockIdx.x ;
   //  printf("Input %d, thread %d\n", input[index].x, threadIdx.x);  
   for(int i=0; i<n ; i++){
      // if add input[index] doms input[i]
      if( dom(input[index],input[i] ) ){
         output[i] = 80085; // test
         
         //  printf("Input %d, thread %d\n", input[index].x, threadIdx.x);  
      }    
   }
}
__global__ 
void name_maker(const char *input, int *output){
   // int index = threadIdx.x + blockIdx.x*(1024);
   int index = threadIdx.x + blockIdx.x;

   // printf("In GPU section\n");
   
   // if(output[threadIdx.x + blockIdx.x*(1024)] == 0){
   if(output[index] == 0){
      int R_INDEX = threadIdx.x*4 + blockIdx.x;
      // printf("out==0\n");
      printf("name %c%c%c%c INDEX %d BLOCK ID %d\n"
      ,input[R_INDEX],input[R_INDEX+1],input[R_INDEX+2],input[R_INDEX+3], index, blockIdx.x);
   }

}


int main()
{
   int N = sizeof(data_array)/ sizeof(XY);
   // std::cout<<"size N: "<<N<<"\n"; 

   char *de_char_names;
   hipMalloc((void **) &de_char_names, (4*N)*sizeof(char)); //5 becasue of ending \0


   XY *de_input;
   hipMalloc((void **) &de_input, N*sizeof(XY));


   int *de_counter;
   hipMalloc((void **) &de_counter, N*sizeof(int));

   auto const start_time = std::chrono::system_clock::now();


   hipMemcpy( de_input, &data_array, sizeof(XY)*N, hipMemcpyHostToDevice );
   
   // std::cout<<"size of(xy)" << sizeof(de_input);
   
   int result[ N ];
   for(int i =0; i<N;i++){
      result[i]=0;
   }

   hipMemcpy( de_counter, &result, sizeof(result), hipMemcpyHostToDevice );

   hipMemcpy( de_char_names, char_data, (4*N)*sizeof(char), hipMemcpyHostToDevice );

   //block, threads
   //TEST
   // 1024
   // solv<<<N, 1>>>(N, de_input, de_counter);
   solv<<<N, 1>>>(N, de_input, de_counter);
   //block X threads = N



   // Wait for GPU to finish before accessing on host
   hipDeviceSynchronize();

   //TEST
   name_maker<<<N, 1>>>(de_char_names, de_counter);


   hipDeviceSynchronize();// probably dont need this


   // Once the kernel has completed, we initiate a transfer of the result data *back to the CPU*.
   // Note that the `hipMemcpyDeviceToHost` constant denotes transferring data *from the GPU*.
   hipMemcpy( result, de_counter, N*sizeof(int), hipMemcpyDeviceToHost );


   auto const end_time = std::chrono::system_clock::now();
   auto const elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time );
   std::cout << "time: " << ( elapsed_time.count() ) << " us" << std::endl;
   
   //  Generate Final Result
   std::cout<<"____________________________________________________-";

   // Free memory
   hipFree(de_input);
   hipFree(de_counter);

   //TEST
   hipFree(de_char_names);
   
   return 0; 
}



