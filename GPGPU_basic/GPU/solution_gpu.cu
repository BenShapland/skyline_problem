#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>     // timing library
#include <numeric>    // std::accumulate()

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// #include "data-sanity-check.hpp"
#include "test-data.hpp"

#define N 5000 // num_blocks * num_thread_per_block 
#define num_blocks  5
#define num_thread_per_block  1000 //2048


__constant__  XY de_data_array[N];


__host__ __device__
bool dom(XY a, XY b){
   if( ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y))){
      return true; 
   }
   return false;
}

__global__ 
void name_maker(const char *input, int *output){
   int index = threadIdx.x + (blockIdx.x* num_thread_per_block);

 
   if(output[index] != 80085){
      int R_INDEX = threadIdx.x*4 + (blockIdx.x* num_thread_per_block *4);
      printf("%c%c%c%c\n"
      ,input[R_INDEX],input[R_INDEX+1],input[R_INDEX+2],input[R_INDEX+3]);
   }

}

__global__ 
void solv( int * counter )
{
   int index = threadIdx.x + (blockIdx.x * num_thread_per_block);

   for(int i=0; i<N ; i++){

      if( dom(de_data_array[i],de_data_array[index] ) ){
         counter[index] = 80085; // test
      }    
   }

}


int main()
{
 
 
   
   hipMemcpyToSymbol(HIP_SYMBOL(de_data_array), &data_array, N*sizeof(XY));  //Save to constant


   int *de_counter;
   hipMalloc((void **) &de_counter, N*sizeof(int));
   

   char *de_char_names;
   hipMalloc((void **) &de_char_names, (4*N)*sizeof(char)); 


   int result[ N ];

   hipMemcpy( de_counter, &result, sizeof(result), hipMemcpyHostToDevice );
   
   //copy names
   hipMemcpy( de_char_names, char_data, (4*N)*sizeof(char), hipMemcpyHostToDevice );
   
   auto const start_time = std::chrono::system_clock::now();


   solv<<<num_blocks, num_thread_per_block>>>(de_counter);
   hipDeviceSynchronize();

   
   name_maker<<<num_blocks, num_thread_per_block>>>(de_char_names, de_counter);
   hipDeviceSynchronize();

   auto const end_time = std::chrono::system_clock::now();
   auto const elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time );
   std::cout << "time: " << ( elapsed_time.count() ) << " us" << std::endl;
   
   hipMemcpy( result, de_counter, N*sizeof(int), hipMemcpyDeviceToHost );

   //free
   hipFree(de_counter);
   hipFree(de_char_names);

   return 0; 
}



