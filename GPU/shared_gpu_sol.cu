#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>     // timing library
#include <numeric>    // std::accumulate()
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "data/data.cpp"
// #include "data/data-sanity-check.hpp"
// #include "data/test-data.hpp"


#define N 500000
#define num_blocks 500
#define num_thread_per_block 1000


__device__  XY de_data_array[N];

//dom returns True if a dominates b else false
__host__ __device__
bool dom(XY a, XY b)
{
   bool a_better =false;
   a_better = ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y));
   return a_better;

}


__global__ 
void name_maker(const char *input, int *output)
{
   int index = threadIdx.x + (blockIdx.x* num_thread_per_block);
   if(output[index] != 80085){
      int R_INDEX = threadIdx.x*4 + (blockIdx.x* num_thread_per_block *4);
      printf("%c%c%c%c "
      ,input[R_INDEX],input[R_INDEX+1],input[R_INDEX+2],input[R_INDEX+3]);
   }

}

// test a single block                                                 
__global__ void one_stencil (int *counter, int n)
{
   

   __shared__ XY s[num_thread_per_block];   
   int localId = threadIdx.x;
   int globalId;
   int index = localId + num_thread_per_block*blockIdx.x;
   XY MY_NODE = de_data_array[index];

   for(int i =0; i< num_blocks; i++){
      // copy
      globalId = localId + num_thread_per_block*i;
      s[localId] = de_data_array[globalId];
      __syncthreads();
      // solv
      for(int k=0;k<num_thread_per_block;k++){
         if( dom(s[k],MY_NODE) ){

            counter[index] = 80085;
            // return;
            for(int j =i; j<num_blocks;j++){
                  __syncthreads();
                  globalId = localId + num_thread_per_block*j;
                  s[localId] = de_data_array[globalId];
                  __syncthreads();
            }
            return;
         }
      }
      // sync
      __syncthreads();
   }

}


int instance(int seed)
{

   XY data_array[N];
   xy_data_gen(data_array,N,seed);
   
   char char_data[N*4];
   char_array_gen(char_data,N,seed);
   // Copy to Global
   hipMemcpyToSymbol(HIP_SYMBOL(de_data_array), &data_array, N*sizeof(XY));  
   
   // To store from solve
   int *de_counter;
   hipMalloc((void **) &de_counter, N*sizeof(int));
   

   char *de_char_names;
   hipMalloc((void **) &de_char_names, (4*N)*sizeof(char)); 

   hipMemcpy( de_char_names, char_data, (4*N)*sizeof(char), hipMemcpyHostToDevice );
   

   // START
   auto const start_time = std::chrono::system_clock::now();

   one_stencil<<<num_blocks, num_thread_per_block>>>(de_counter,N);
   hipDeviceSynchronize();
   name_maker<<<num_blocks, num_thread_per_block>>>(de_char_names, de_counter);
   hipDeviceSynchronize();

   // END
   auto const end_time = std::chrono::system_clock::now();
   auto const elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time );
   // std::cout << "time: " << ( elapsed_time.count() ) << " us" << std::endl;
   
   //free
   hipFree(de_counter);
   hipFree(de_char_names);

   return elapsed_time.count(); 
}

int main(int argc, char** argv)
{
   int seed = 1;
   int number_of_runs = 1000;
   
   int total_time=0;
   if(argc < 2 ){
      std::cout<<"Mode: compare or bench\n";
      return 0;
   }

   std::string input = argv[1];  
   if(argc==2 || argc==3){
      if(argc==3){
         std::istringstream a2(argv[2]);
         a2>>number_of_runs;
         seed = number_of_runs;;
      }  
      if (input == "bench" ){
         printf("----------BENCHMARKING-----------\n");
         // BENCHMARKING
         std::cout<<"num of data_points: "<<N<<"\n";
         std::cout<<"num of test runs: "<<number_of_runs<<"\n";    
         for(auto i=0; i <number_of_runs; i++){
            std::srand( (i *i*10000)/7 +4 );
            seed = std::rand(); 
            total_time += instance(seed);
            std::cout<<"\n";
         }
         int avg_time = total_time/number_of_runs;
         std::cout<<"avg_time: "<< avg_time <<" \n";
      
      }else if(input == "compare"){
         printf("----------COMPARISON-----------\n");
         std::cout<<"num of data_points: "<<N<<"\n";  
         std::cout<<"using seed: "<<seed<<"\n";
         total_time += instance(seed);
         std::cout<<"\n";
      }else{
         std::cout<<"Mode: compare or bench\n";
      }
   }
   else std::cout<<"Incorrect Number of Arguments\n";
   return 0;

}




