
/**
 * Toy example porting a simd algorithm to CUDA for an NVIDIA GPU
 *
 * Calculates the average size of set of 3d vectors.
 * Example input: {{1,1,1},{2,2,2}}
 * I.e., 1 vector of size (3*1^2)^-0.5 and 1 vector of size (3*2^2)^0.5
 * Example output: ( sqrt(3) + sqrt(12) ) / 2 = 2.59808
 */
 
#include <hip/hip_runtime.h>
#include <iostream>
 #include <vector>
 
//  #include "Node.h"
//  #include "test-data.hpp"
 
 
 #include <random> 	// std::rand, std::srand, std::default_random_engine
 std::string random_string(std::size_t length)
 {
 
     const std::string characters = "abcdefghijklmnopqrstuvwxyz";
 
     std::random_device random_device;
     std::mt19937 generator(random_device());
     std::uniform_int_distribution<> distribution(0, characters.size() - 1);
 
     std::string random_string;
 
     for (std::size_t i = 0; i < length; ++i)
     {
         random_string += characters[distribution(generator)];
     }
 
     return random_string;
 }
 
 
 
 std::vector<std::string> make_names(int number_of_names){
    std::vector<std::string> hold;
    for(int i =0; i<number_of_names; i++){
       hold.push_back(random_string(4));
    }
    return hold;
 }
 
 
 
 __global__ 
 void solv(int n, int *input, int *output)
 {
   output[0] = 69;
 }
 
 
 
 
 int main()
 {
    //  make test data NODE


    int test_data[10] = { 1 , 2, 3, 4, 5, 6, 7, 8, 9 ,10};
    
 
    std::vector <std::string> name = make_names(10);
     
    //test
    std::cout<< (test_data[0]) <<"\n";std::cout<< (test_data[0]) <<"\n";std::cout<< (name[0]) << "\n";
 
 
    int N = (10);
 
    std::cout<< "N: " <<N << "\n";//10
 
    // allocate memmory
    int *de_input;
    hipMalloc((void **) &de_input, N*sizeof(int));
 
    int *de_counter;
    hipMalloc((void **) &de_counter, N*sizeof(int));
 
    //stop
    hipMemcpy( de_input, test_data, sizeof(test_data), hipMemcpyHostToDevice );
 
 
    //block, threads
    solv<<<1, 1>>>(N, de_input, de_counter);
    //block X threads = N
 
 
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
 
 
    int result[ N ];
    result[0] = 99;
    // Once the kernel has completed, we initiate a transfer of the result data *back to the CPU*.
    // Note that the `cudaMemcpyDeviceToHost` constant denotes transferring data *from the GPU*.
    hipMemcpy( result, de_counter, sizeof(de_counter), hipMemcpyDeviceToHost );
 
 
 
 
    std::cout<< "RESULTS: "<< (result[0])<< "\n";
 
 
    // Free memory
    hipFree(de_input);
    hipFree(de_counter);
 
    return 0; 
 }