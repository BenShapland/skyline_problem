#include "hip/hip_runtime.h"

/**
 * Toy example porting a simd algorithm to CUDA for an NVIDIA GPU
 *
 * Calculates the average size of set of 3d vectors.
 * Example input: {{1,1,1},{2,2,2}}
 * I.e., 1 vector of size (3*1^2)^-0.5 and 1 vector of size (3*2^2)^0.5
 * Example output: ( sqrt(3) + sqrt(12) ) / 2 = 2.59808
 */
#include <iostream>
#include <vector>

#include "Node.h"
#include "test-data.hpp"


#include <random> 	// std::rand, std::srand, std::default_random_engine
 



__host__ __device__
bool dom(XY a, XY b){
   if( ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y))){
       return true;
   }
   return false;
}

__global__ 
void solv(int n, XY *input, int *output)
{
// Printing input
   int index = threadIdx.x;
   printf("Input %d, thread %d\n", input[index].x, threadIdx.x); 




   // int index = threadIdx.x;
   // for(int i=0; i<n ; i++){
   //    // if add input[index] doms input[i]
   //    if( dom(input[index],input[i] ) ){
   //       output[i] = 80085; // test
         
   //       printf("Input %d, thread %d\n", input[index].x, threadIdx.x);  
   //    }
      
   
   // }
   
}


int main()
{
   //  make test data NODE
   Node test_data;
   test_data.xy = data;
   XY * data_pointer = data;
   test_data.name = make_names(10);
    
   int test_array[10] = {0,1,2,3,4,5,6,7,8,9};


   //test
   std::cout<< (test_data.xy[0].x) <<"\n";std::cout<< (test_data.xy[0].y) <<"\n";std::cout<< (test_data.name[0]) << "\n";


   int N = (test_data.xy.size()  );

   std::cout<< "N: " <<N << "\n";//10

   // allocate memmory
   // std::vector < XY > *de_input;
   XY *de_input;


   hipMalloc((void **) &de_input, N*sizeof(XY));

   int *de_counter;
   hipMalloc((void **) &de_counter, N*sizeof(int));


   hipMalloc((void **) &de_input, N*sizeof(XY));




   //stop
   hipMemcpy( de_input, &data_pointer, sizeof(XY)*N, hipMemcpyHostToDevice );
   
   // std::cout<<"size of(xy)" << sizeof(de_input);
   
   int result[ N ];
   for (int i =0;i<N;i++){
      result[i] = 999;
   }
   hipMemcpy( de_counter, &result, sizeof(result), hipMemcpyHostToDevice );


   //block, threads
   solv<<<1, N>>>(N, de_input, de_counter);
   //block X threads = N


   // Wait for GPU to finish before accessing on host
   hipDeviceSynchronize();


   // int result[ N ];
   // for (int i =0;i<N;i++){
   //    result[i] = 999;
   // }

   // Once the kernel has completed, we initiate a transfer of the result data *back to the CPU*.
   // Note that the `hipMemcpyDeviceToHost` constant denotes transferring data *from the GPU*.
   hipMemcpy( result, de_counter, N*sizeof(int), hipMemcpyDeviceToHost );


   for (int i =0;i<N;i++){
      std::cout<< i<<" " <<(result[i])<< "\n";
   }
   
   //goal 0 , 1, 4, 7, 14


   // std::cout<< "0: "<< (result[0])<< "\n";
   // std::cout<< "1: "<< (result[1])<< "\n";
   // std::cout<< "2: "<< (result[2])<< "\n";
   // std::cout<< "3: "<< (result[3])<< "\n";
   // std::cout<< "4: "<< (result[4])<< "\n";

   // Free memory
   hipFree(de_input);
   hipFree(de_counter);

   return 0; 
}