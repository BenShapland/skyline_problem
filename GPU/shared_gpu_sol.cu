#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>     // timing library
#include <numeric>    // std::accumulate()


#include <iostream> // for outputting (printing) to streams (e.g., the terminal)
#include <random> 	// std::rand, std::srand, std::default_random_engine
#include <cmath>	// sqrt()
#include <cstring>   // std::strcmp()
#include <fstream>   // std::ofstream
#include <stdlib.h>  // srand(), rand()
#include <algorithm> // std::sort(), std::for_each()
#include <vector>
#include <sstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// #include "Node.h"
#include "data/data.cpp"

// #include "data/data-sanity-check.hpp"
// #include "data/test-data.hpp"


#define N 60000 // num_blocks * num_thread_per_block 
#define num_blocks 60
#define num_thread_per_block  1000 //2048


__device__  XY de_data_array[N];


//dom returns True if a dominates b else false
__host__ __device__
bool dom(XY a, XY b){
   bool a_better =false;
   a_better = ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y));
   return a_better;

}


__global__ 
void name_maker(const char *input, int *output){
   int index = threadIdx.x + (blockIdx.x* num_thread_per_block);

 
   if(output[index] != 80085){
      int R_INDEX = threadIdx.x*4 + (blockIdx.x* num_thread_per_block *4);
      printf("%c%c%c%c, "
      ,input[R_INDEX],input[R_INDEX+1],input[R_INDEX+2],input[R_INDEX+3]);
      printf("(%d,%d) \n", de_data_array[index].x,de_data_array[index].y);
   }

}

// test a single block                                                 
__global__ void one_stencil (int *counter, int n)
{
   
   // extern __shared__ XY s[];
   __shared__ XY s[num_thread_per_block];
   
   int localId = threadIdx.x;

   int globalId;

   int index = localId + num_thread_per_block*blockIdx.x;

   XY MY_NODE = de_data_array[index];

   // MIGHT WANT TO CLEAN UP I so that blocks dont just copy from the same location over and over again;

   for(int i =0; i< num_blocks; i++){
      // copy
      globalId = localId + num_thread_per_block*i;
      s[localId] = de_data_array[globalId];
      // sync - can it just be threads in a block
      __syncthreads();
      // solv
      for(int k=0;k<num_thread_per_block;k++){
         if( dom(s[k],MY_NODE) ){

            counter[index] = 80085;
            // return;
            for(int j =i; j<num_blocks;j++){
                  __syncthreads();
                  globalId = localId + num_thread_per_block*j;
                  s[localId] = de_data_array[globalId];
                  __syncthreads();
            }
            return;
         }
      }
      // sync
      __syncthreads();
   }

}

__global__ 
void solv( int * counter )
{
   int index = threadIdx.x + (blockIdx.x * num_thread_per_block);

   for(int i=0; i<N ; i++){

      if( dom(de_data_array[i],de_data_array[index] ) ){
         counter[index] = 80085;
         return;  // added this line
      }    
   }

}



int main(int argc, char** argv){
    
   int seed=1;   // default value
   if(argc ==2){
      std::istringstream a1(argv[1]);
      a1>>seed;
   }
  
   XY data_array[N];
   xy_data_gen(data_array,N,seed);
   
   char char_data[N*4];
   char_array_gen(char_data,N,seed);
   // Copy to Global
   hipMemcpyToSymbol(HIP_SYMBOL(de_data_array), &data_array, N*sizeof(XY));  
   
   // To store from solve
   int *de_counter;
   hipMalloc((void **) &de_counter, N*sizeof(int));
   

   char *de_char_names;
   hipMalloc((void **) &de_char_names, (4*N)*sizeof(char)); 

   hipMemcpy( de_char_names, char_data, (4*N)*sizeof(char), hipMemcpyHostToDevice );
   

   // START
   auto const start_time = std::chrono::system_clock::now();
   
   one_stencil<<<num_blocks, num_thread_per_block>>>(de_counter,N);
   // solv<<<num_blocks, num_thread_per_block>>>(de_counter);
   hipDeviceSynchronize();

   // device to host
   // hipMemcpy( result, de_counter, N*sizeof(int), hipMemcpyDeviceToHost );
  
   name_maker<<<num_blocks, num_thread_per_block>>>(de_char_names, de_counter);
   hipDeviceSynchronize();


   // END
   auto const end_time = std::chrono::system_clock::now();
   auto const elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time );
   std::cout << "time: " << ( elapsed_time.count() ) << " us" << std::endl;
   
   //free
   hipFree(de_counter);
   hipFree(de_char_names);

   return 0; 
}



