#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>     // timing library
#include <numeric>    // std::accumulate()
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>



#include "data/data-sanity-check.hpp"
// #include "data/test-data.hpp"
// #include "data/data.cpp"

#define N 16 // num_blocks * num_thread_per_block 
#define num_blocks 1
#define num_thread_per_block  16 //2048


// #define N 600000 // num_blocks * num_thread_per_block 
// #define num_blocks 600
// #define num_thread_per_block  1000 //2048


// __constant__  XY de_data_array[N];
__device__  XY de_data_array[N]; // DATA XY values

__device__  int manhatten_array[N]; // Manhatten distance

__device__ int original_index[N]; // original index




//dom returns True if a dominates b else false
__host__ __device__
bool dom(XY a, XY b){
   bool a_better =false;
   a_better = ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y));
   return a_better;

}


__host__ __device__ bool operator<(XY a, XY b) { 
   if(a.x == b.x){
      if(a.y == b.y) return true;
   } 
   return false;

};

__host__ __device__
bool distance_sort(XY a, XY b){
   if(a.x == b.x){
      if(a.y == b.y) return true;
   } 
   return false;

}


__global__
void print_it_all(){

   for(int i =0 ; i < N ; i++){
      printf("%d %d\n",de_data_array[i].x,de_data_array[i].y);
   }
}





// __global__
// void wtf( XY *data){
//    for(int i =0 ; i < N ; i++){
//       printf("%d %d\n",de_data_array[i].x,de_data_array[i].y);
//    }

//    thrust::sort(data,data+N,distance_sort);

//    for(int i =0 ; i < N ; i++){
//       printf(gpu:
	nvcc -O3  solution_gpu.cu -o gpu 

   "%d %d\n",de_data_array[i].x,de_data_array[i].y);
//    }
// }




int main()


{
  
   // int seed = 1;
   // XY data_array[N];
   // xy_data_gen(data_array,N,seed);
   
   // char char_data[N*4];
   // char_array_gen(char_data,N,seed);

   hipMemcpyToSymbol(HIP_SYMBOL(de_data_array), &data_array, N*sizeof(XY));  //Save to global

   
   XY * test;
   hipMalloc((void **) &test, N*sizeof(XY));
   hipMemcpy(test,data_array,N*sizeof(XY),hipMemcpyHostToDevice);

   wtf<<<1,1>>>(test);
   hipDeviceSynchronize();

   // thrust::sort(test,test+N,distance_sort);




   // void* data_ptr = 0;
   // hipGetSymbolAddress(&data_ptr, HIP_SYMBOL(&data_array));
   // thrust::device_ptr<XY> data_first(reinterpret_cast<XY*>(&data_ptr));

  

   // thrust::sort(data_first,data_first+N,distance_sort);
   // hipDeviceSynchronize();

   // print_it_all<<<1,1>>>();
   // hipDeviceSynchronize();







   // char *de_char_names;
   // hipMalloc((void **) &de_char_names, (4*N)*sizeof(char)); 

   // hipMemcpy( de_char_names, char_data, (4*N)*sizeof(char), hipMemcpyHostToDevice );
   
   
   // // START
   // auto const start_time = std::chrono::system_clock::now();

   // // Make manhatten distance array
   // MakeManhattan <<<num_blocks,num_thread_per_block >>>();
   // hipDeviceSynchronize();

   // // Setup pointers to adresses in device for thrust sort
   // void* array_ptr = 0;
   // hipGetSymbolAddress(&array_ptr, HIP_SYMBOL(manhatten_array));
   // thrust::device_ptr<int> manhattan_first(reinterpret_cast<int*>(array_ptr));
   
   // // MAKE COPY OF MANHATEN for sorting again
   // thrust::device_vector<int> HOLD (manhattan_first,manhattan_first+N);
   // hipDeviceSynchronize();

   // void* data_ptr = 0;
   // hipGetSymbolAddress(&data_ptr, HIP_SYMBOL(de_data_array));
   // thrust::device_ptr<XY> data_first(reinterpret_cast<XY*>(data_ptr));
   
   // // Sort DATA based on manhatten distance array
   // thrust::sort_by_key(manhattan_first, manhattan_first + N, data_first);
   // // Sort the original indexs based on HOLD (unsorted manhatten)
   // void* index_ptr = 0;
   // hipGetSymbolAddress(&index_ptr, HIP_SYMBOL(original_index));
   // thrust::device_ptr<int> index_first(reinterpret_cast<int*>(index_ptr));

   // thrust::sort_by_key(HOLD.begin(), HOLD.end(), index_first);
   // hipDeviceSynchronize();


   // test <<< 1,1 >>>(de_char_names);
   // // print_it_all<<<1,1>>>();
   // hipDeviceSynchronize();


   // // END
   // auto const end_time = std::chrono::system_clock::now();
   // auto const elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time );
   // std::cout << "time: " << ( elapsed_time.count() ) << " us" << std::endl;
   

   return 0; 
}



