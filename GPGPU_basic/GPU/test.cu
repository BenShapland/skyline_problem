#include "hip/hip_runtime.h"

/**
 * Toy example porting a simd algorithm to CUDA for an NVIDIA GPU
 *
 * Calculates the average size of set of 3d vectors.
 * Example input: {{1,1,1},{2,2,2}}
 * I.e., 1 vector of size (3*1^2)^-0.5 and 1 vector of size (3*2^2)^0.5
 * Example output: ( sqrt(3) + sqrt(12) ) / 2 = 2.59808
 */
 #include <iostream>
 #include <vector>
 #include <chrono>     // timing library
#include <numeric>    // std::accumulate()
 
//  #include "Node.h"
 #include "test-data.hpp"
 
 
 #include <random> 	// std::rand, std::srand, std::default_random_engine
 std::string random_string(std::size_t length)
 {
     const std::string characters = "abcdefghijklmnopqrstuvwxyz";
 
     std::random_device random_device;
     std::mt19937 generator(random_device());
     std::uniform_int_distribution<> distribution(0, characters.size() - 1);
 
     std::string random_string;
 
     for (std::size_t i = 0; i < length; ++i)
     {
         random_string += characters[distribution(generator)];
     }
 
     return random_string;
 }
 
 
 std::vector<std::string> make_names(int number_of_names){
    std::vector<std::string> hold;
    for(int i =0; i<number_of_names; i++){
       hold.push_back(random_string(4));
    }
    return hold;
 }
 

 __host__ __device__
 bool dom(XY a, XY b){
    if( ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y))){
        return true; 
    }
    return false;
 }
 
 __global__ 
 void solv(int n, XY *input, int *output)
 {
 // Printing input
    int index = threadIdx.x;
   //  printf("Input %d, thread %d\n", input[index].x, threadIdx.x);  
    for(int i=0; i<n ; i++){
       // if add input[index] doms input[i]
       if( dom(input[index],input[i] ) ){
          output[i] = 80085; // test
          
         //  printf("Input %d, thread %d\n", input[index].x, threadIdx.x);  
       }    
    }
 }
 
 
 int main()
 {



   int N = sizeof(data_array)/ sizeof(XY);

   // std::cout<<"size N: "<<N<<"\n";
 
    XY *de_input;
    hipMalloc((void **) &de_input, N*sizeof(XY));


    int *de_counter;
    hipMalloc((void **) &de_counter, N*sizeof(int));

   auto const start_time = std::chrono::system_clock::now();


    hipMemcpy( de_input, &data_array, sizeof(XY)*N, hipMemcpyHostToDevice );
    
    // std::cout<<"size of(xy)" << sizeof(de_input);
    
    int result[ N ];
   //  for (int i =0;i<N;i++){
   //     result[i] = 0;
   //  }
    hipMemcpy( de_counter, &result, sizeof(result), hipMemcpyHostToDevice );
 
 
    //block, threads
    solv<<<1, N>>>(N, de_input, de_counter);
    //block X threads = N
 
 
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
 
 
    // int result[ N ];
    // for (int i =0;i<N;i++){
    //    result[i] = 999;
    // }
 
    // Once the kernel has completed, we initiate a transfer of the result data *back to the CPU*.
    // Note that the `hipMemcpyDeviceToHost` constant denotes transferring data *from the GPU*.
    hipMemcpy( result, de_counter, N*sizeof(int), hipMemcpyDeviceToHost );
 

    // Generate Final Result
   Node final_result;
   for (int i =0;i<N;i++){
      // std::cout<< i<<" " <<(result[i])<< "\n";
      if (result[i] == 0){
         // final_result.add(XY_array.x,XY_array.y,name[i]);
         std::cout<<"x: " << data_array[i].x <<" y: "<<data_array[i].y<<"\n";
      }

   }
      
    //goal 0 , 1, 4, 7, 14
      
 
    // Free memory
    hipFree(de_input);
    hipFree(de_counter);

    auto const end_time = std::chrono::system_clock::now();
    auto const elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time );
    std::cout << "time: " << ( elapsed_time.count() ) << " us" << std::endl;
    return 0; 
 }