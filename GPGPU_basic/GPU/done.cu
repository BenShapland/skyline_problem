#include "hip/hip_runtime.h"

/**
 * Toy example porting a simd algorithm to CUDA for an NVIDIA GPU
 *
 * Calculates the average size of set of 3d vectors.
 * Example input: {{1,1,1},{2,2,2}}
 * I.e., 1 vector of size (3*1^2)^-0.5 and 1 vector of size (3*2^2)^0.5
 * Example output: ( sqrt(3) + sqrt(12) ) / 2 = 2.59808
 */
#include <iostream>
#include <vector>

#include "Node.h"
#include "test-data.hpp"


#include <random> 	// std::rand, std::srand, std::default_random_engine
std::string random_string(std::size_t length)
{

    const std::string characters = "abcdefghijklmnopqrstuvwxyz";

    std::random_device random_device;
    std::mt19937 generator(random_device());
    std::uniform_int_distribution<> distribution(0, characters.size() - 1);

    std::string random_string;

    for (std::size_t i = 0; i < length; ++i)
    {
        random_string += characters[distribution(generator)];
    }

    return random_string;
}



std::vector<std::string> make_names(int number_of_names){
   std::vector<std::string> hold;
   for(int i =0; i<number_of_names; i++){
      hold.push_back(random_string(4));
   }
   return hold;
}



__global__ 
void solv(int n, std::vector <XY> *input, int *output)
{
  output[0] = 69;
}




int main()
{
   //  make test data NODE
   Node test_data;
   test_data.xy = data;

   test_data.name = make_names(10);
    
   //test
   std::cout<< (test_data.xy[0].x) <<"\n";std::cout<< (test_data.xy[0].y) <<"\n";std::cout<< (test_data.name[0]) << "\n";


   int N = (test_data.xy.size()  );

   std::cout<< "N: " <<N << "\n";//10

   // allocate memmory
   std::vector < XY > *de_input;
   hipMalloc((void **) &de_input, N*sizeof(XY));

   int *de_counter;
   hipMalloc((void **) &de_counter, N*sizeof(int));

   //stop
   hipMemcpy( de_input, &test_data.xy, sizeof(test_data.xy), hipMemcpyHostToDevice );


   //block, threads
   solv<<<1, 1>>>(N, de_input, de_counter);
   //block X threads = N


   // Wait for GPU to finish before accessing on host
   hipDeviceSynchronize();


   int result[ N ];
   result[0] = 99;

   // Once the kernel has completed, we initiate a transfer of the result data *back to the CPU*.
   // Note that the `hipMemcpyDeviceToHost` constant denotes transferring data *from the GPU*.
   hipMemcpy( result, de_counter, sizeof(de_counter), hipMemcpyDeviceToHost );




   std::cout<< "RESULTS: "<< (result[0])<< "\n";


   // Free memory
   hipFree(de_input);
   hipFree(de_counter);

   return 0; 
}