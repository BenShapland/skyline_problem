#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>     // timing library
#include <numeric>    // std::accumulate()
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "data/data.cpp"
// #include "data/data-sanity-check.hpp"
// #include "data/test-data.hpp"


#define N 5000
#define num_blocks 5
#define num_thread_per_block 1000


__device__  XY de_data_array[N];

//dom returns True if a dominates b else false
__host__ __device__
bool dom(XY a, XY b)
{
   bool a_better =false;
   a_better = ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y));
   return a_better;

}


__global__ 
void name_maker(const char *input, int *output)
{
   int index = threadIdx.x + (blockIdx.x* num_thread_per_block);
   if(output[index] != 80085){
      int R_INDEX = threadIdx.x*4 + (blockIdx.x* num_thread_per_block *4);
      printf("%c%c%c%c\n"
      ,input[R_INDEX],input[R_INDEX+1],input[R_INDEX+2],input[R_INDEX+3]);
   }

}

// test a single block                                                 
__global__ void one_stencil (int *counter, int n)
{
   

   __shared__ XY s[num_thread_per_block];   
   int localId = threadIdx.x;
   int globalId;
   int index = localId + num_thread_per_block*blockIdx.x;
   XY MY_NODE = de_data_array[index];

   for(int i =0; i< num_blocks; i++){
      // copy
      globalId = localId + num_thread_per_block*i;
      s[localId] = de_data_array[globalId];
      __syncthreads();
      // solv
      for(int k=0;k<num_thread_per_block;k++){
         if( dom(s[k],MY_NODE) ){

            counter[index] = 80085;
            // return;
            for(int j =i; j<num_blocks;j++){
                  __syncthreads();
                  globalId = localId + num_thread_per_block*j;
                  s[localId] = de_data_array[globalId];
                  __syncthreads();
            }
            return;
         }
      }
      // sync
      __syncthreads();
   }

}


int main(int argc, char** argv)
{
   int seed=1;   // default value
   if(argc ==2){
      std::istringstream a1(argv[1]);
      a1>>seed;
   }
  
   XY data_array[N];
   xy_data_gen(data_array,N,seed);
   
   char char_data[N*4];
   char_array_gen(char_data,N,seed);
   // Copy to Global
   hipMemcpyToSymbol(HIP_SYMBOL(de_data_array), &data_array, N*sizeof(XY));  
   
   // To store from solve
   int *de_counter;
   hipMalloc((void **) &de_counter, N*sizeof(int));
   

   char *de_char_names;
   hipMalloc((void **) &de_char_names, (4*N)*sizeof(char)); 

   hipMemcpy( de_char_names, char_data, (4*N)*sizeof(char), hipMemcpyHostToDevice );
   

   // START
   auto const start_time = std::chrono::system_clock::now();
   one_stencil<<<num_blocks, num_thread_per_block>>>(de_counter,N);
   hipDeviceSynchronize();
   name_maker<<<num_blocks, num_thread_per_block>>>(de_char_names, de_counter);
   hipDeviceSynchronize();


   // END
   auto const end_time = std::chrono::system_clock::now();
   auto const elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time );
   std::cout << "time: " << ( elapsed_time.count() ) << " us" << std::endl;
   
   //free
   hipFree(de_counter);
   hipFree(de_char_names);

   return 0; 
}



