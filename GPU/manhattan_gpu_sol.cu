#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>     // timing library
#include <numeric>    // std::accumulate()
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "data/data.cpp"
// #include "data/data-sanity-check.hpp"
// #include "data/test-data.hpp"

#define N 5000
#define num_blocks 5
#define num_thread_per_block 1000


__device__  XY de_data_array[N]; // DATA XY values
__device__  int manhatten_array[N]; // Manhatten distance
__device__ int original_index[N]; // original index


//dom returns True if a dominates b else false
__host__ __device__
bool dom(XY a, XY b)
{
   bool a_better =false;
   a_better = ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y));
   return a_better;
}


// spawn a thread for every node 
// cacluate the manhaten distance and store to an array 
__global__
void MakeManhattan()
{
   int index = threadIdx.x + (blockIdx.x * num_thread_per_block);
   int man = de_data_array[index].x + de_data_array[index].y;
   // printf("at index %d : %d\n",index,man);
   manhatten_array[index] = man;
   original_index[index] = index;
   // printf("FROM ARRAY at index %d : %d\n",index, manhatten_array[index]);
}

__global__
void compare_D_to_batch(int index_of_batch,int *de_index){
  // line 6-12 compare each node in D to the first batch size.
   //if a point is dominated it is added to a list of dominated points
   //and the thread returns

   int index = threadIdx.x + (blockIdx.x * num_thread_per_block);

   // printf("index[%d] threadID %d BlockID %d\n",index,threadIdx.x,blockIdx.x);
   //compare to global
   // if dom mark it 
   // if im dom return
   for(int i =index_of_batch; i < N; i++){
      // if im domed return
      if(de_data_array[index].x == 80085){
         return;
      }
      // if i dom some node set it to 80085
      if(dom(de_data_array[index],de_data_array[i])){
         de_data_array[i].x = 80085;
      }

   }
   if(de_data_array[index].x != 80085){
      // printf("original index: %d with (%d,%d)\n",original_index[index],de_data_array[index].x,de_data_array[index].y);
      de_index[index] = original_index[index];
   }

}




int main(int argc, char** argv){

   int seed=1;   // default value
   if(argc ==2){
      std::istringstream a1(argv[1]);
      a1>>seed;
   }

   XY data_array[N];
   xy_data_gen(data_array,N,seed);
   
   char char_data[N*4];
   char_array_gen(char_data,N,seed);


   // Set-up 
   //--------------------------------------------------------------------------------------

   hipMemcpyToSymbol(HIP_SYMBOL(de_data_array), &data_array, N*sizeof(XY));  //Save to global
   
   // Make manhatten distance array
   MakeManhattan <<<num_blocks,num_thread_per_block >>>();
   hipDeviceSynchronize();

   // Setup pointers to adresses in device for thrust sort
   void* array_ptr = 0;
   hipGetSymbolAddress(&array_ptr, HIP_SYMBOL(manhatten_array));
   thrust::device_ptr<int> manhattan_first(reinterpret_cast<int*>(array_ptr));
   
   // MAKE COPY OF MANHATEN for sorting again
   thrust::device_vector<int> HOLD (manhattan_first,manhattan_first+N);
   hipDeviceSynchronize();

   void* data_ptr = 0;
   hipGetSymbolAddress(&data_ptr, HIP_SYMBOL(de_data_array));
   thrust::device_ptr<XY> data_first(reinterpret_cast<XY*>(data_ptr));
   
   // Sort DATA based on manhatten distance array
   thrust::sort_by_key(manhattan_first, manhattan_first + N, data_first);
   // Sort the original indexs based on HOLD (unsorted manhatten)
   void* index_ptr = 0;
   hipGetSymbolAddress(&index_ptr, HIP_SYMBOL(original_index));
   thrust::device_ptr<int> index_first(reinterpret_cast<int*>(index_ptr));

   thrust::sort_by_key(HOLD.begin(), HOLD.end(), index_first);
   hipDeviceSynchronize();

   // hipFree(index_first);
   // free thrust hold???

   //--------------------------------------------------------------------------------------


   /*
      line 6-12 compare each node in D to the first batch size.
      if a point is dominated it is added to a list of dominated points
      and the thread returns
   */

   // size of batch
   int size_of_batch = N;  // MIGHT WANT TO CHANGE THIS UP A BIT
   int result[ size_of_batch ];
   for(int i=0; i<size_of_batch;i++){
      result[i] = -1;
   }

   int *de_counter;
   hipMalloc((void **) &de_counter, N*sizeof(int));


   // hipMemcpy( de_index, &batch_index, sizeof(int)*size_of_batch, hipMemcpyHostToDevice );

   hipMemcpy( de_counter, &result, sizeof(result), hipMemcpyHostToDevice );


   int starting_place =0;
   
   // START
   auto const start_time = std::chrono::system_clock::now();
   
   // for(){
   compare_D_to_batch <<< 2,8 >>>(starting_place,de_counter);
   hipDeviceSynchronize();

   // THIS DOES NOT WORK@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@2
   // @@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
   // @@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
   // Could try and copy over soltion block by block

   hipMemcpy( result, de_counter, N*sizeof(int), hipMemcpyDeviceToHost ); 

   hipDeviceSynchronize();

   for(int k = 0; k < size_of_batch; k++ ){
      if (result[k] != -1){
         int dex = result[k]*4;
         printf("%c%c%c%c\n", 
         char_data[dex],
         char_data[dex+1],
         char_data[dex+2],
         char_data[dex+3]
         );
      }
   }

   // END
   auto const end_time = std::chrono::system_clock::now();
   auto const elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time );
   std::cout << "time: " << ( elapsed_time.count() ) << " us" << std::endl;

   hipFree(de_counter);


   // }
   //line 13 now that all threads have finished. Remove the dominated points from D


   //line 14-15 record the NON-dominated points and remove them from D.
   //(MIGHT BE ABLE TO RETURN THEM HERE ASWELL)

   //THIS IS ALL LOOPED THROUGH UNTILL THE DATA SET BECOMES 0\
   //each time we loop we will romove batch size number of points
   //except for the last run
   


      // FREE SHIT PLEASE


   return 0; 
}



