#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>     // timing library
#include <numeric>    // std::accumulate()
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "data/data.cpp"
// #include "data/data-sanity-check.hpp"
// #include "data/test-data.hpp"

#define N 5000
#define num_blocks 5
#define num_thread_per_block 1000

__constant__  XY de_data_array[N];

__host__ __device__
bool dom(XY a, XY b){
   if( ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y))){
      return true; 
   }
   return false;
}


// //dom returns True if a dominates b else false
// __host__ __device__
// bool dom(XY a, XY b)
// {
//    bool a_better =false;
//    a_better = ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y));
//    return a_better;

// }


__global__ 
void name_maker(const char *input, int *output)
{
   int index = threadIdx.x + (blockIdx.x* num_thread_per_block); 
   if(output[index] != 80085){
      int R_INDEX = threadIdx.x*4 + (blockIdx.x* num_thread_per_block *4);
      printf("%c%c%c%c "
      ,input[R_INDEX],input[R_INDEX+1],input[R_INDEX+2],input[R_INDEX+3]);
   }

}

__global__ 
void solv( int * counter )
{
   int index = threadIdx.x + (blockIdx.x * num_thread_per_block);

   for(int i=0; i<N ; i++){

      if( dom(de_data_array[i],de_data_array[index] ) ){
         counter[index] = 80085;
         return;  // added this line
      }    
   }
}

int instance(int seed)
{
   
   XY data_array[N];
   xy_data_gen(data_array,N,seed);
   char char_data[N*4];
   char_array_gen(char_data,N,seed);

   
   hipMemcpyToSymbol(HIP_SYMBOL(de_data_array), &data_array, N*sizeof(XY));  //Save to constant


   int *de_counter;
   hipMalloc((void **) &de_counter, N*sizeof(int));
   

   char *de_char_names;
   hipMalloc((void **) &de_char_names, (4*N)*sizeof(char)); 


   
   //copy names
   hipMemcpy( de_char_names, char_data, (4*N)*sizeof(char), hipMemcpyHostToDevice );
   
   // Start timer
   auto const start_time = std::chrono::system_clock::now();

   solv<<<num_blocks, num_thread_per_block>>>(de_counter);
   hipDeviceSynchronize();
  
   name_maker<<<num_blocks, num_thread_per_block>>>(de_char_names, de_counter);
   hipDeviceSynchronize();

   // End timer 
   auto const end_time = std::chrono::system_clock::now();
   auto const elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time );
   // std::cout << "time: " << ( elapsed_time.count() ) << " us" << std::endl;


   //free
   hipFree(de_counter);
   hipFree(de_char_names);

   return  elapsed_time.count(); 
}


int main(int argc, char** argv)
{
   int seed = 1;
   int number_of_runs = 1000;
   
   int total_time=0;
   if(argc < 2 ){
      std::cout<<"Mode: compare or bench\n";

      return 0;
   }

   std::string input = argv[1];
   if(argc==2 || argc==3){
      if(argc==3){
         std::istringstream a2(argv[2]);
         a2>>number_of_runs;
         seed = number_of_runs;;
      }  
      if (input == "bench" ){
         printf("----------BENCHMARKING-----------\n");
         // BENCHMARKING
         std::cout<<"num of data_points: "<<N<<"\n";
         std::cout<<"num of test runs: "<<number_of_runs<<"\n";    
         for(auto i=0; i <number_of_runs; i++){
            std::srand( (i *i*10000)/7 +4 );
            seed = std::rand(); 
            total_time += instance(seed);
            std::cout<<"\n";
         }
         int avg_time = total_time/number_of_runs;
         std::cout<<"avg_time: "<< avg_time <<" \n";
      
      }else if(input == "compare"){
         printf("----------COMPARISON-----------\n");
         std::cout<<"num of data_points: "<<N<<"\n";  
         std::cout<<"using seed: "<<seed<<"\n";
         total_time += instance(seed);
         std::cout<<"\n";
      }else{
         std::cout<<"Mode: compare or bench\n";

      }
   }
   else std::cout<<"Incorrect Number of Arguments\n";
   return 0;

}


