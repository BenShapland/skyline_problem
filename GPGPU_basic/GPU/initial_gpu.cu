#include "hip/hip_runtime.h"

#include <iostream>
#include <vector>
#include <chrono>     // timing library
#include <numeric>    // std::accumulate()

#include "test-data.hpp"
// #include "data-sanity-check.hpp"




__host__ __device__
bool dom(XY a, XY b){
   if( ((a.x < b.x)&&(a.y <= b.y)) || ((a.x <= b.x)&&(a.y < b.y))){
      return true; 
   }
   return false;
}


__global__ 
void solv(int n, XY *input, int *output)
{

   int index = threadIdx.x + blockIdx.x ;
   for(int i=0; i<n ; i++){
      if( dom(input[index],input[i] ) ){
         output[i] = 80085; // test
         
      }    
   }
}
__global__ 
void name_maker(const char *input, int *output){
   int index = threadIdx.x + blockIdx.x;


   if(output[index] == 0){
      int R_INDEX = threadIdx.x*4 + blockIdx.x;
      // printf("out==0\n");
      printf("name %c%c%c%c INDEX %d BLOCK ID %d\n"
      ,input[R_INDEX],input[R_INDEX+1],input[R_INDEX+2],input[R_INDEX+3], index, blockIdx.x);
   }

}


int main()
{
   int N = sizeof(data_array)/ sizeof(XY);

   char *de_char_names;
   hipMalloc((void **) &de_char_names, (4*N)*sizeof(char)); 


   XY *de_input;
   hipMalloc((void **) &de_input, N*sizeof(XY));


   int *de_counter;
   hipMalloc((void **) &de_counter, N*sizeof(int));

   auto const start_time = std::chrono::system_clock::now();


   hipMemcpy( de_input, &data_array, sizeof(XY)*N, hipMemcpyHostToDevice );
   
   
   int result[ N ];
   for(int i =0; i<N;i++){
      result[i]=0;
   }

   hipMemcpy( de_counter, &result, sizeof(result), hipMemcpyHostToDevice );

   hipMemcpy( de_char_names, char_data, (4*N)*sizeof(char), hipMemcpyHostToDevice );

;
   solv<<<1, N>>>(N, de_input, de_counter);
   //block X threads = N



   // Wait for GPU to finish before accessing on host
   hipDeviceSynchronize();

   name_maker<<<1, N>>>(de_char_names, de_counter);


   hipDeviceSynchronize();// probably dont need this


   // Once the kernel has completed, we initiate a transfer of the result data *back to the CPU*.
   // Note that the `hipMemcpyDeviceToHost` constant denotes transferring data *from the GPU*.
   hipMemcpy( result, de_counter, N*sizeof(int), hipMemcpyDeviceToHost );


   auto const end_time = std::chrono::system_clock::now();
   auto const elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time );
   std::cout << "time: " << ( elapsed_time.count() ) << " us" << std::endl;
   

   hipFree(de_input);
   hipFree(de_counter);
   hipFree(de_char_names);
   
   return 0; 
}



